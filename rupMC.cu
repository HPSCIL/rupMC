#include "rupMC.h"

#include "MCTriangleTable.h"
#include "MC.h"

#include "hip/hip_runtime.h"
#include ""

#include <time.h>
#include <iostream>

#include <string>
#include <fstream>

int rupMC(int myid, float value, int readloop, int *Time, int *dims, double *origins, double *spacing, float *data, int *Loops, float*** PTS, int*** TRA, int** count, int* PTSTRAStartStop)
{
	time_t t[2] = { 0 };

	int C;
	hipGetDeviceCount(&C);
	int GPUId = (myid - 2) % C;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(GPUId);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPUId);
	unsigned int globalmem = deviceProp.totalGlobalMem;

	size_t PerKSize = (dims[0] * dims[1] * (sizeof(float) + 4 * sizeof(int)) + (dims[0] - 1)*(dims[1] - 1)*(5 * 3 * 3 * sizeof(float) + 5 * 3 * sizeof(int)))*0.8;
	size_t PerStep = (globalmem - (3 + 8 + 256 * 16 + 12) * sizeof(int) - sizeof(float)) / PerKSize;
	*Loops = 1;
	if (PerStep < dims[2])
	{
		*Loops = dims[2] / PerStep + 1;
		PerStep = dims[2] / *Loops + 3;
	}

	*PTS = new float*[*Loops];
	*TRA = new int*[*Loops];
	*count = new int[*Loops * 2]{ 0 };
	int *d_dims;
	hipMalloc((void**)&d_dims, 3 * sizeof(int));
	hipMemcpy(d_dims, dims, 3 * sizeof(int), hipMemcpyHostToDevice);
	double *d_spacing, *d_origin;
	hipMalloc((void**)&d_spacing, 3 * sizeof(double));
	hipMemcpy(d_spacing, spacing, 3 * sizeof(double), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_origin, 3 * sizeof(double));
	hipMemcpy(d_origin, origins, 3 * sizeof(double), hipMemcpyHostToDevice);
	int *d_CASE_MASK, *d_MC_TRIANGLE_TABLE;
	hipMalloc((void**)&d_CASE_MASK, 8 * sizeof(int));
	hipMemcpy(d_CASE_MASK, CASE_MASK, 8 * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_MC_TRIANGLE_TABLE, 256 * 16 * sizeof(int));
	hipMemcpy(d_MC_TRIANGLE_TABLE, MC_TRIANGLE_TABLE, 256 * 16 * sizeof(int), hipMemcpyHostToDevice);
	int edgeToIdx[12];
	edgeToIdx[0] = 0;
	edgeToIdx[1] = 4;
	edgeToIdx[2] = dims[0] * 3;
	edgeToIdx[3] = 1;
	edgeToIdx[4] = dims[0] * dims[1] * 3;
	edgeToIdx[5] = dims[0] * dims[1] * 3 + 4;
	edgeToIdx[6] = (dims[0] * dims[1] + dims[0]) * 3;
	edgeToIdx[7] = dims[0] * dims[1] * 3 + 1;
	edgeToIdx[8] = 2;
	edgeToIdx[9] = 5;
	edgeToIdx[10] = dims[0] * 3 + 2;
	edgeToIdx[11] = (dims[0] + 1) * 3 + 2;
	int *d_edgeToIdx;
	hipMalloc((void**)&d_edgeToIdx, 12 * sizeof(int));
	hipMemcpy(d_edgeToIdx, edgeToIdx, 12 * sizeof(int), hipMemcpyHostToDevice);
	int PTSStartIndex = 0;

	if ((myid - 2) == 0)
	{
		PTSTRAStartStop[0] = 0;
		PTSTRAStartStop[2] = 0;
	}
	if ((myid - 2) == readloop - 1)
	{
		PTSTRAStartStop[1] = 1;
		PTSTRAStartStop[3] = 1;
	}

	for (int L = 0, StartK = 0; L < *Loops&&StartK < dims[2]; L++, StartK += PerStep - 3)
	{
		if (L == *Loops - 1)
			PerStep = dims[2] - StartK;

		int blkwidth = 16;
		int blkheight = 16;
		dim3 blocks(blkwidth, blkheight);
		dim3 grids((dims[0] - 1) % blkwidth == 0 ? (dims[0] - 1) / blkwidth : (dims[0] - 1) / blkwidth + 1, (dims[1] - 1) % blkheight == 0 ? (dims[1] - 1) / blkheight : (dims[1] - 1) / blkheight + 1);

		hipMemcpy(&d_dims[2], &PerStep, sizeof(int), hipMemcpyHostToDevice);
		float *d_data;
		hipMalloc((void**)&d_data, sizeof(float) * dims[0] * dims[1] * PerStep);
		hipMemcpy(d_data, &data[StartK*dims[0] * dims[1]], sizeof(float) * dims[0] * dims[1] * PerStep, hipMemcpyHostToDevice);

		int *d_PTSMark;
		hipMalloc((void**)&d_PTSMark, (dims[0] * dims[1] * PerStep) * 3 * sizeof(int));
		hipMemset(d_PTSMark, 0, (dims[0] * dims[1] * PerStep) * 3 * sizeof(int));
		int *d_TRAMark;
		hipMalloc((void**)&d_TRAMark, (dims[0] * dims[1] * PerStep) * sizeof(int));
		hipMemset(d_TRAMark, 0, (dims[0] * dims[1] * PerStep) * sizeof(int));

		time(&t[0]);
		MCComputeMark << <grids, blocks >> > (d_dims, d_data, value, d_CASE_MASK, d_MC_TRIANGLE_TABLE, d_edgeToIdx, d_PTSMark, d_TRAMark);
		hipDeviceSynchronize();
		time(&t[1]);
		*Time += t[1] - t[0];
		
		int* PTSMark = new int[dims[0] * dims[1] * PerStep * 3];
		int* TRAMark = new int[dims[0] * dims[1] * PerStep];
		hipMemcpy(PTSMark, d_PTSMark, (dims[0] * dims[1] * PerStep) * 3 * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(TRAMark, d_TRAMark, (dims[0] * dims[1] * PerStep) * sizeof(int), hipMemcpyDeviceToHost);
		int MarkStart = dims[0] * dims[1];
		int MarkStop = (PerStep - 1)*dims[0] * dims[1];
		if (L == 0)
			MarkStart = 0;
		if (L == *Loops - 1)
			MarkStop = PerStep*dims[0] * dims[1];
		for (int i = MarkStart * 3 + 1; i < MarkStop * 3; i++)
			PTSMark[i] += PTSMark[i - 1];
		for (int i = MarkStart + 1; i < MarkStop; i++)
			TRAMark[i] += TRAMark[i - 1];
		count[0][L * 2] = PTSMark[(MarkStop - dims[0] * dims[1]) * 3 - 1];
		count[0][L * 2 + 1] = TRAMark[(MarkStop - dims[0] * dims[1]) - 1];

		if (PTSTRAStartStop[0] != 0 && L == 0)
		{
			PTSTRAStartStop[0] = PTSMark[PTSTRAStartStop[0] * dims[0] * dims[1] * 3 - 1];
			PTSTRAStartStop[2] = TRAMark[PTSTRAStartStop[2] * dims[0] * dims[1] - 1];
		}
		if (L == *Loops - 1)
		{
			PTSTRAStartStop[1] = PTSMark[(MarkStop - PTSTRAStartStop[1] * dims[0] * dims[1]) * 3 - 1];
			PTSTRAStartStop[3] = TRAMark[(MarkStop - PTSTRAStartStop[3] * dims[0] * dims[1]) - 1];
		}

		int tempdims = PerStep - 2;
		if (L > 0 && L < *Loops - 1)
			tempdims = PerStep - 3;
		if (L == *Loops - 1)
			tempdims = PerStep - 1;
		hipMemcpy(&d_dims[2], &tempdims, sizeof(int), hipMemcpyHostToDevice);
		double origin = origins[2] + (StartK + MarkStart / (dims[0] * dims[1])) *spacing[2];
		hipMemcpy(&d_origin[2], &origin, sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(d_PTSMark, &PTSMark[MarkStart * 3], (MarkStop - MarkStart) * 3 * sizeof(int), hipMemcpyHostToDevice);
		float *d_PTS;
		hipMalloc((void**)&d_PTS, count[0][L * 2] * 3 * sizeof(float));
		hipMemset(d_PTS, 0, count[0][L * 2] * 3 * sizeof(float));

		time(&t[0]);
		MCComputePoint << <grids, blocks >> > (d_dims, d_spacing, d_origin, &d_data[MarkStart],value, d_PTSMark, d_PTS);
		hipDeviceSynchronize();
		time(&t[1]);
		*Time += t[1] - t[0];

		PTS[0][L] = new float[count[0][L * 2] * 3];
		hipMemcpy(PTS[0][L], d_PTS, count[0][L * 2] * 3 * sizeof(float), hipMemcpyDeviceToHost);

		tempdims = PerStep;
		if (L > 0 && L < *Loops - 1)
			tempdims = PerStep - 2;
		if (L == *Loops - 1)
			tempdims = PerStep - 1;
		hipMemcpy(&d_dims[2], &tempdims, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_TRAMark, &TRAMark[MarkStart], (MarkStop - MarkStart) * sizeof(int), hipMemcpyHostToDevice);
		int *d_TRA;
		hipMalloc((void**)&d_TRA, count[0][L * 2 + 1] * 3 * sizeof(int));
		hipMemset(d_TRA, 0, count[0][L * 2 + 1] * 3 * sizeof(int));

		time(&t[0]);
		MCComputeTriangle << <grids, blocks >> > (d_dims, &d_data[MarkStart], value,d_CASE_MASK, d_MC_TRIANGLE_TABLE, d_edgeToIdx, d_PTSMark, d_TRAMark, d_TRA, PTSStartIndex);
		hipDeviceSynchronize();
		time(&t[1]);
		*Time += t[1] - t[0];

		TRA[0][L] = new int[count[0][L * 2 + 1] * 3];
		hipMemcpy(TRA[0][L], d_TRA, count[0][L * 2 + 1] * 3 * sizeof(int), hipMemcpyDeviceToHost);
		PTSStartIndex += count[0][L * 2];

		hipFree(d_data);
		hipFree(d_PTSMark);
		hipFree(d_TRAMark);
		hipFree(d_PTS);
		hipFree(d_TRA);

		delete[] PTSMark;
		delete[] TRAMark;
	}
	hipFree(d_dims);
	hipFree(d_spacing);
	hipFree(d_origin);
	hipFree(d_CASE_MASK);
	hipFree(d_MC_TRIANGLE_TABLE);
	hipFree(d_edgeToIdx);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}


